#include "../sha256.h"
#include "common.h"

static unsigned long long cpucycles(void) {
    unsigned long long result;
    __asm volatile(".byte 15;.byte 49;shlq $32,%%rdx;orq %%rdx,%%rax" : "=a"(result)::"%rdx");

    return result;
} // cpucycles

void sha2_speed_test();
void sha2_validity_test();

int main(int argc, char** argv) {
    sha2_speed_test();
    sha2_validity_test();

    return 0;
} // main

void sha2_speed_test() {
    struct timespec start, stop;
    double result;

    u64 hash_msg_bytes = 1024 * 1024 * 1024; // whole data

    hash_msg_bytes *= 16; // 16 GB
    u32 msg_num = 1024 * 1024;
    u8 *d, *md, *gpu_md, *gpu_para_md;

    CHECK(hipHostMalloc(&d, hash_msg_bytes));
    CHECK(hipHostMalloc(&md, 32));
    CHECK(hipHostMalloc(&gpu_md, 32));
    CHECK(hipHostMalloc(&gpu_para_md, 32 * msg_num));
    for (u64 i = 0; i < hash_msg_bytes; i++)
        d[i] = 2;

    printf("\nsha256 speed test\n");

    printf("-------------------CPU test--------------------\n");
    for (int i = 0; i < 10; i++) // warm up
        sha256(md, d, 1024);
    for (int i = 1; i < 20; i++) {
        int msg_size = (2 << i);
        clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
        sha256(md, d, msg_size);
        clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
        result = (stop.tv_sec - start.tv_sec) * 1e6 + (stop.tv_nsec - start.tv_nsec) / 1e3;
        printf("cpu: %dB \t%10.2lf us\t%10.2lfMB/s\n", msg_size, result, msg_size / result);
    }

    printf("\n");
    printf("---------------gpu one core test----------------\n");
    face_sha256(gpu_md, d, 1024, 1000);
    int iter = 100;
    for (int i = 1; i < 20; i++) {
        int msg_size = (2 << i);
        face_sha256(gpu_md, d, msg_size, iter);
        printf("gpu: %dB \t%10.2lf us\t%10.2lfMB/s\n", msg_size, g_result / iter,
               msg_size / g_result * iter);
    }

    // printf("\n");
    // printf("---------------gpu dp test (82 * 512)----------------\n");
    // msg_num = 82 * 512;
    // for (int i = 1; ; i++) {
    // 	int msg_size = (2 << i);
    // 	if ((u64)msg_size * msg_num > hash_msg_bytes) break;
    // 	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
    // 	t0 = cpucycles();
    // 	face_dp_sha256((const u8 *)d, gpu_para_md, msg_size,
    // 		       msg_num, 82, 512);
    // 	t1 = cpucycles();
    // 	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
    // 	result = (stop.tv_sec - start.tv_sec) * 1e6
    // 		 + (stop.tv_nsec - start.tv_nsec) / 1e3;
    // 	printf("pra %d B, \t%.2lf us\t%.2lfMB/s\tcycles: %lld\tC/B: %.2lf\n",
    // 	       msg_size, result, (double)msg_size * msg_num / result,
    // 	       t1 - t0, (t1 - t0) * 1.0 / msg_size / msg_num);
    // }
    //
    // printf("\n");
    // printf("---------------gpu msdp test (82 * 512)----------------\n");
    // msg_num = 82 * 512;
    // for (int i = 10; ; i++) {
    // 	int msg_size = (2 << i);
    // 	if ((u64)msg_size * msg_num > hash_msg_bytes) break;
    // 	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
    // 	t0 = cpucycles();
    // 	face_msdp_sha256((const u8 *)d, gpu_para_md, msg_size,
    // 			 msg_num, 82, 512);
    // 	t1 = cpucycles();
    // 	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
    // 	result = (stop.tv_sec - start.tv_sec) * 1e6
    // 		 + (stop.tv_nsec - start.tv_nsec) / 1e3;
    // 	printf("pra %d B, \t%.2lf us\t%.2lfMB/s\tcycles: %lld\tC/B: %.2lf\n",
    // 	       msg_size, result, (double)msg_size * msg_num / result,
    // 	       t1 - t0, (t1 - t0) * 1.0 / msg_size / msg_num);
    // }

    /*
            printf("\n");
            printf("---------------gpu dp test (82 * 1024 * 8)----------------\n");
            msg_num = 82 * 1024 * 8; // should < 1024 * 1024
            for (int i = 1; ; i++) {
                    int msg_size = (2 << i);
                    if ((u64)msg_size * msg_num > hash_msg_bytes) break;
                    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
                    t0 = cpucycles();
                    face_dp_sha256((const u8 *)d, gpu_para_md, msg_size,
                                   msg_num, msg_num / 32, 32);
                    t1 = cpucycles();
                    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
                    result = (stop.tv_sec - start.tv_sec) * 1e6
     + (stop.tv_nsec - start.tv_nsec) / 1e3;
                    printf("pra %d B, \t%.2lf us\t%.2lfMB/s\tcycles: %lld\tC/B: %.2lf\n",
                           msg_size, result, (double)msg_size * msg_num / result,
                           t1 - t0, (t1 - t0) * 1.0 / msg_size / msg_num);
            }

            printf("\n");
            printf("---------------gpu dp test (128 * 1024 * 8)----------------\n");
            msg_num = 128 * 1024 * 8; // should < 1024 * 1024
            for (int i = 1; ; i++) {
                    int msg_size = (2 << i);
                    if ((u64)msg_size * msg_num > hash_msg_bytes) break;
                    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
                    t0 = cpucycles();
                    face_dp_sha256((const u8 *)d, gpu_para_md, msg_size,
                                   msg_num, msg_num / 32, 32);
                    t1 = cpucycles();
                    clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
                    result = (stop.tv_sec - start.tv_sec) * 1e6
     + (stop.tv_nsec - start.tv_nsec) / 1e3;
                    printf("pra %d B, \t%.2lf us\t%.2lfMB/s\tcycles: %lld\tC/B: %.2lf\n",
                           msg_size, result, (double)msg_size * msg_num / result,
                           t1 - t0, (t1 - t0) * 1.0 / msg_size / msg_num);
            }
     */
} // sha2_speed_test

void sha2_validity_test() {
    struct timespec start, stop;
    double result;

    u32 se_msg_B = 1024 * 1024;
    u32 s_msg_B = 32;                 // single message size
    u32 p_msg_B = 82 * 512 * s_msg_B; // for parallel test
    u32 msg_N = p_msg_B / s_msg_B;

    printf("msg_N = %d\n", msg_N);
    u8 *d, *cpu_md, *gpu_md, *cpu_para_md, *gpu_para_md;

    int right;

    CHECK(hipHostMalloc(&d, p_msg_B));
    CHECK(hipHostMalloc(&cpu_md, 32));
    CHECK(hipHostMalloc(&gpu_md, 32));
    CHECK(hipHostMalloc(&cpu_para_md, 32 * msg_N));
    CHECK(hipHostMalloc(&gpu_para_md, 32 * msg_N));
    for (int i = 0; i < p_msg_B; i++)
        d[i] = i;
    for (int i = 0; i < p_msg_B; i += 7)
        d[i] += i;

    printf("\nsha256 test\n");
    cout << flush;

    sha256(cpu_md, d, se_msg_B);
    face_sha256(gpu_md, d, se_msg_B, 1);

    right = 1;
    for (int j = 0; j < 32; j++) {
        if (cpu_md[j] != gpu_md[j]) {
            right = 0;
            break;
        }
    }
    if (right == 1)
        printf("single core check right!\n");
    else
        printf("single core check wrong!\n");

    // /* parallel test */
    // for (int j = 0; j < msg_N; j++) {
    // 	self_SHA256((const u8 *)(d + j * s_msg_B), s_msg_B, cpu_para_md + j * 32);
    // }
    //
    // clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
    // t0 = cpucycles();
    // face_dp_sha256((const u8 *)d, gpu_para_md, s_msg_B,
    // 	       msg_N, 82, 512);
    // t1 = cpucycles();
    // clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
    // result = (stop.tv_sec - start.tv_sec) * 1e6
    // 	 + (stop.tv_nsec - start.tv_nsec) / 1e3;
    // printf("ex pra sha256\t%.2lf us\t%.2lfMB/s\tcycles: %lld\tC/B: %.2lf\n",
    //        result, p_msg_B / result, t1 - t0, (t1 - t0) * 1.0 / p_msg_B);
    //
    // right = 1;
    // for (int j = 0; j < 32; j++) {
    // 	if (cpu_para_md[j] != gpu_para_md[j]) {
    // 		right = 0;
    // 		break;
    // 	}
    // }
    // if (right == 1) printf("parallel check right!\n");
    // else printf("parallel check wrong!\n");
    //
    // /* multi stream test
    //  * When s_msg_B is small, the effect is obvious
    //  */
    // clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
    // t0 = cpucycles();
    // face_msdp_sha256((const u8 *)d, gpu_para_md, s_msg_B,
    // 		 msg_N, 82, 512);
    // t1 = cpucycles();
    // clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);
    // result = (stop.tv_sec - start.tv_sec) * 1e6
    // 	 + (stop.tv_nsec - start.tv_nsec) / 1e3;
    // printf("ex pra sha256\t%.2lf us\t%.2lfMB/s\tcycles: %lld\tC/B: %.2lf\n",
    //        result, p_msg_B / result, t1 - t0, (t1 - t0) * 1.0 / p_msg_B);
    //
    // right = 1;
    // for (int j = 0; j < 32; j++) {
    // 	if (cpu_para_md[j] != gpu_para_md[j]) {
    // 		right = 0;
    // 		break;
    // 	}
    // }
    // if (right == 1) printf("multi stream check right!\n");
    // else printf("multi stream check wrong!\n");

} // sha2_validity_test
