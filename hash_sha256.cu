#include "hip/hip_runtime.h"


#include <stdint.h>
#include <string.h>
#include "common.h"
#include <iostream>
using namespace std;

#include "address.h"
#include "utils.h"
#include "params.h"
#include "hash.h"
#include "sha256.h"

#ifdef SHA256

/* For SHA256, there is no immediate reason to initialize at the start,
   so this function is an empty operation. */
void initialize_hash_function(const unsigned char *pub_seed,
			      const unsigned char *sk_seed)
{
	seed_state(pub_seed);
	(void)sk_seed; /* Suppress an 'unused parameter' warning. */
} // initialize_hash_function

__device__ void dev_initialize_hash_function(const unsigned char *pub_seed,
					     const unsigned char *sk_seed)
{
	dev_seed_state(pub_seed);
	(void)sk_seed; /* Suppress an 'unused parameter' warning. */
} // dev_initialize_hash_function

__global__ void global_initialize_hash_function(const unsigned char *pub_seed,
						const unsigned char *sk_seed)
{
	dev_initialize_hash_function(pub_seed, sk_seed);
} // dev_initialize_hash_function

void face_initialize_hash_function(const unsigned char *pub_seed,
				   const unsigned char *sk_seed)
{
	int device = DEVICE_USED;
	u8 *dev_pub_seed = NULL, *dev_sk_seed = NULL;

	CHECK(hipSetDevice(device));

	CHECK(hipMalloc((void **)&dev_pub_seed, SPX_N * sizeof(u8)));
	CHECK(hipMemcpy(dev_pub_seed, pub_seed, SPX_N * sizeof(u8), HOST_2_DEVICE));

	CHECK(hipMalloc((void **)&dev_sk_seed, SPX_N * sizeof(u8)));
	CHECK(hipMemcpy(dev_sk_seed, sk_seed, SPX_N * sizeof(u8), HOST_2_DEVICE));

	CHECK(hipDeviceSynchronize());
	global_initialize_hash_function << < 1, 1 >> >
		(dev_pub_seed, dev_sk_seed);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());

	hipFree(dev_pub_seed); hipFree(dev_sk_seed);
} // face_initialize_hash_function

/*
 * Computes PRF(key, addr), given a secret key of SPX_N bytes and an address
 */
void prf_addr(unsigned char *out, const unsigned char *key,
	      const uint32_t addr[8])
{
	unsigned char buf[SPX_N + SPX_SHA256_ADDR_BYTES];
	unsigned char outbuf[SPX_SHA256_OUTPUT_BYTES];

	memcpy(buf, key, SPX_N);
	memcpy(buf + SPX_N, addr, SPX_SHA256_ADDR_BYTES);

	sha256(outbuf, buf, SPX_N + SPX_SHA256_ADDR_BYTES);
	memcpy(out, outbuf, SPX_N);
} // prf_addr

__device__ void dev_prf_addr(unsigned char *out, const unsigned char *key,
			     const uint32_t addr[8])
{
	unsigned char buf[SPX_N + SPX_SHA256_ADDR_BYTES];
	unsigned char outbuf[SPX_SHA256_OUTPUT_BYTES];

	memcpy(buf, key, SPX_N);
	memcpy(buf + SPX_N, addr, SPX_SHA256_ADDR_BYTES);

	dev_sha256(outbuf, buf, SPX_N + SPX_SHA256_ADDR_BYTES);
	memcpy(out, outbuf, SPX_N);
} // dev_prf_addr

/**
 * Computes the message-dependent randomness R, using a secret seed as a key
 * for HMAC, and an optional randomization value prefixed to the message.
 * This requires m to have at least SPX_SHA256_BLOCK_BYTES + SPX_N space
 * available in front of the pointer, i.e. before the message to use for the
 * prefix. This is necessary to prevent having to move the message around (and
 * allocate memory for it).
 */
void gen_message_random(unsigned char *R, const unsigned char *sk_prf,
			const unsigned char *optrand,
			const unsigned char *m, unsigned long long mlen)
{
	unsigned char buf[SPX_SHA256_BLOCK_BYTES + SPX_SHA256_OUTPUT_BYTES];
	uint8_t state[40];
	int i;

#if SPX_N > SPX_SHA256_BLOCK_BYTES
    #error "Currently only supports SPX_N of at most SPX_SHA256_BLOCK_BYTES"
#endif // if SPX_N > SPX_SHA256_BLOCK_BYTES

	/* This implements HMAC-SHA256 */
	for (i = 0; i < SPX_N; i++) {
		buf[i] = 0x36 ^ sk_prf[i];
	}
	memset(buf + SPX_N, 0x36, SPX_SHA256_BLOCK_BYTES - SPX_N);

	sha256_inc_init(state);
	sha256_inc_blocks(state, buf, 1);

	memcpy(buf, optrand, SPX_N);

	/* If optrand + message cannot fill up an entire block */
	if (SPX_N + mlen < SPX_SHA256_BLOCK_BYTES) {
		memcpy(buf + SPX_N, m, mlen);
		sha256_inc_finalize(buf + SPX_SHA256_BLOCK_BYTES, state,
				    buf, mlen + SPX_N);
	}
	/* Otherwise first fill a block, so that finalize only uses the message */
	else {
		memcpy(buf + SPX_N, m, SPX_SHA256_BLOCK_BYTES - SPX_N);
		sha256_inc_blocks(state, buf, 1);

		m += SPX_SHA256_BLOCK_BYTES - SPX_N;
		mlen -= SPX_SHA256_BLOCK_BYTES - SPX_N;
		sha256_inc_finalize(buf + SPX_SHA256_BLOCK_BYTES, state, m, mlen);
	}

	for (i = 0; i < SPX_N; i++) {
		buf[i] = 0x5c ^ sk_prf[i];
	}
	memset(buf + SPX_N, 0x5c, SPX_SHA256_BLOCK_BYTES - SPX_N);

	sha256(buf, buf, SPX_SHA256_BLOCK_BYTES + SPX_SHA256_OUTPUT_BYTES);
	memcpy(R, buf, SPX_N);
} // gen_message_random

__device__ void dev_gen_message_random(unsigned char *R, const unsigned char *sk_prf,
				       const unsigned char *optrand,
				       const unsigned char *m, unsigned long long mlen)
{
	unsigned char buf[SPX_SHA256_BLOCK_BYTES + SPX_SHA256_OUTPUT_BYTES];
	uint8_t state[40];
	int i;

#if SPX_N > SPX_SHA256_BLOCK_BYTES
    #error "Currently only supports SPX_N of at most SPX_SHA256_BLOCK_BYTES"
#endif // if SPX_N > SPX_SHA256_BLOCK_BYTES

	/* This implements HMAC-SHA256 */
	for (i = 0; i < SPX_N; i++) {
		buf[i] = 0x36 ^ sk_prf[i];
	}
	memset(buf + SPX_N, 0x36, SPX_SHA256_BLOCK_BYTES - SPX_N);

	dev_sha256_inc_init(state);
	dev_sha256_inc_blocks(state, buf, 1);

	memcpy(buf, optrand, SPX_N);

	/* If optrand + message cannot fill up an entire block */
	if (SPX_N + mlen < SPX_SHA256_BLOCK_BYTES) {
		memcpy(buf + SPX_N, m, mlen);
		dev_sha256_inc_finalize(buf + SPX_SHA256_BLOCK_BYTES, state,
					buf, mlen + SPX_N);
	}
	/* Otherwise first fill a block, so that finalize only uses the message */
	else {
		memcpy(buf + SPX_N, m, SPX_SHA256_BLOCK_BYTES - SPX_N);
		dev_sha256_inc_blocks(state, buf, 1);

		m += SPX_SHA256_BLOCK_BYTES - SPX_N;
		mlen -= SPX_SHA256_BLOCK_BYTES - SPX_N;
		dev_sha256_inc_finalize(buf + SPX_SHA256_BLOCK_BYTES, state, (void *)m, mlen);
	}

	for (i = 0; i < SPX_N; i++) {
		buf[i] = 0x5c ^ sk_prf[i];
	}
	memset(buf + SPX_N, 0x5c, SPX_SHA256_BLOCK_BYTES - SPX_N);

	dev_sha256(buf, buf, SPX_SHA256_BLOCK_BYTES + SPX_SHA256_OUTPUT_BYTES);
	memcpy(R, buf, SPX_N);
} // dev_gen_message_random

/**
 * Computes the message hash using R, the public key, and the message.
 * Outputs the message digest and the index of the leaf. The index is split in
 * the tree index and the leaf index, for convenient copying to an address.
 */
void hash_message(unsigned char *digest, uint64_t *tree, uint32_t *leaf_idx,
		  const unsigned char *R, const unsigned char *pk,
		  const unsigned char *m, unsigned long long mlen)
{
#define SPX_TREE_BITS (SPX_TREE_HEIGHT * (SPX_D - 1))
#define SPX_TREE_BYTES ((SPX_TREE_BITS + 7) / 8)
#define SPX_LEAF_BITS SPX_TREE_HEIGHT
#define SPX_LEAF_BYTES ((SPX_LEAF_BITS + 7) / 8)
#define SPX_DGST_BYTES (SPX_FORS_MSG_BYTES + SPX_TREE_BYTES + SPX_LEAF_BYTES)

	unsigned char seed[SPX_SHA256_OUTPUT_BYTES];

	/* Round to nearest multiple of SPX_SHA256_BLOCK_BYTES */
#if (SPX_SHA256_BLOCK_BYTES & (SPX_SHA256_BLOCK_BYTES - 1)) != 0
    #error "Assumes that SPX_SHA256_BLOCK_BYTES is a power of 2"
#endif // if (SPX_SHA256_BLOCK_BYTES & (SPX_SHA256_BLOCK_BYTES - 1)) != 0
#define SPX_INBLOCKS (((SPX_N + SPX_PK_BYTES + SPX_SHA256_BLOCK_BYTES - 1) & \
		       -SPX_SHA256_BLOCK_BYTES) / SPX_SHA256_BLOCK_BYTES)
	unsigned char inbuf[SPX_INBLOCKS * SPX_SHA256_BLOCK_BYTES];

	unsigned char buf[SPX_DGST_BYTES];
	unsigned char *bufp = buf;
	uint8_t state[40];

	sha256_inc_init(state);

	memcpy(inbuf, R, SPX_N);
	memcpy(inbuf + SPX_N, pk, SPX_PK_BYTES);

	/* If R + pk + message cannot fill up an entire block */
	if (SPX_N + SPX_PK_BYTES + mlen < SPX_INBLOCKS * SPX_SHA256_BLOCK_BYTES) {
		memcpy(inbuf + SPX_N + SPX_PK_BYTES, m, mlen);
		sha256_inc_finalize(seed, state, inbuf, SPX_N + SPX_PK_BYTES + mlen);
	}
	/* Otherwise first fill a block, so that finalize only uses the message */
	else {
		memcpy(inbuf + SPX_N + SPX_PK_BYTES, m,
		       SPX_INBLOCKS * SPX_SHA256_BLOCK_BYTES - SPX_N - SPX_PK_BYTES);
		sha256_inc_blocks(state, inbuf, SPX_INBLOCKS);

		m += SPX_INBLOCKS * SPX_SHA256_BLOCK_BYTES - SPX_N - SPX_PK_BYTES;
		mlen -= SPX_INBLOCKS * SPX_SHA256_BLOCK_BYTES - SPX_N - SPX_PK_BYTES;
		sha256_inc_finalize(seed, state, m, mlen);
	}

	/* By doing this in two steps, we prevent hashing the message twice;
	   otherwise each iteration in MGF1 would hash the message again. */
	mgf1(bufp, SPX_DGST_BYTES, seed, SPX_SHA256_OUTPUT_BYTES);

	memcpy(digest, bufp, SPX_FORS_MSG_BYTES);
	bufp += SPX_FORS_MSG_BYTES;

#if SPX_TREE_BITS > 64
    #error For given height and depth, 64 bits cannot represent all subtrees
#endif // if SPX_TREE_BITS > 64

	*tree = bytes_to_ull(bufp, SPX_TREE_BYTES);
	*tree &= (~(uint64_t)0) >> (64 - SPX_TREE_BITS);
	bufp += SPX_TREE_BYTES;

	*leaf_idx = bytes_to_ull(bufp, SPX_LEAF_BYTES);
	*leaf_idx &= (~(uint32_t)0) >> (32 - SPX_LEAF_BITS);
} // hash_message

__device__ void dev_hash_message(unsigned char *digest, uint64_t *tree, uint32_t *leaf_idx,
				 const unsigned char *R, const unsigned char *pk,
				 const unsigned char *m, unsigned long long mlen)
{
#define SPX_TREE_BITS (SPX_TREE_HEIGHT * (SPX_D - 1))
#define SPX_TREE_BYTES ((SPX_TREE_BITS + 7) / 8)
#define SPX_LEAF_BITS SPX_TREE_HEIGHT
#define SPX_LEAF_BYTES ((SPX_LEAF_BITS + 7) / 8)
#define SPX_DGST_BYTES (SPX_FORS_MSG_BYTES + SPX_TREE_BYTES + SPX_LEAF_BYTES)

	unsigned char seed[SPX_SHA256_OUTPUT_BYTES];

	/* Round to nearest multiple of SPX_SHA256_BLOCK_BYTES */
#if (SPX_SHA256_BLOCK_BYTES & (SPX_SHA256_BLOCK_BYTES - 1)) != 0
    #error "Assumes that SPX_SHA256_BLOCK_BYTES is a power of 2"
#endif // if (SPX_SHA256_BLOCK_BYTES & (SPX_SHA256_BLOCK_BYTES - 1)) != 0
#define SPX_INBLOCKS (((SPX_N + SPX_PK_BYTES + SPX_SHA256_BLOCK_BYTES - 1) & \
		       -SPX_SHA256_BLOCK_BYTES) / SPX_SHA256_BLOCK_BYTES)
	unsigned char inbuf[SPX_INBLOCKS * SPX_SHA256_BLOCK_BYTES];

	unsigned char buf[SPX_DGST_BYTES];
	unsigned char *bufp = buf;
	uint8_t state[40];

	dev_sha256_inc_init(state);

	memcpy(inbuf, R, SPX_N);
	memcpy(inbuf + SPX_N, pk, SPX_PK_BYTES);

	/* If R + pk + message cannot fill up an entire block */
	if (SPX_N + SPX_PK_BYTES + mlen < SPX_INBLOCKS * SPX_SHA256_BLOCK_BYTES) {
		memcpy(inbuf + SPX_N + SPX_PK_BYTES, m, mlen);
		dev_sha256_inc_finalize(seed, state, inbuf, SPX_N + SPX_PK_BYTES + mlen);
	}
	/* Otherwise first fill a block, so that finalize only uses the message */
	else {
		memcpy(inbuf + SPX_N + SPX_PK_BYTES, m,
		       SPX_INBLOCKS * SPX_SHA256_BLOCK_BYTES - SPX_N - SPX_PK_BYTES);
		dev_sha256_inc_blocks(state, inbuf, SPX_INBLOCKS);

		m += SPX_INBLOCKS * SPX_SHA256_BLOCK_BYTES - SPX_N - SPX_PK_BYTES;
		mlen -= SPX_INBLOCKS * SPX_SHA256_BLOCK_BYTES - SPX_N - SPX_PK_BYTES;
		dev_sha256_inc_finalize(seed, state, (void *)m, mlen);
	}

	/* By doing this in two steps, we prevent hashing the message twice;
	   otherwise each iteration in MGF1 would hash the message again. */
	dev_mgf1_hg(bufp, SPX_DGST_BYTES, seed, SPX_SHA256_OUTPUT_BYTES);

	memcpy(digest, bufp, SPX_FORS_MSG_BYTES);
	bufp += SPX_FORS_MSG_BYTES;

#if SPX_TREE_BITS > 64
    #error For given height and depth, 64 bits cannot represent all subtrees
#endif // if SPX_TREE_BITS > 64

	*tree = dev_bytes_to_ull(bufp, SPX_TREE_BYTES);
	*tree &= (~(uint64_t)0) >> (64 - SPX_TREE_BITS);
	bufp += SPX_TREE_BYTES;

	*leaf_idx = dev_bytes_to_ull(bufp, SPX_LEAF_BYTES);
	*leaf_idx &= (~(uint32_t)0) >> (32 - SPX_LEAF_BITS);
} // dev_hash_message

#endif
